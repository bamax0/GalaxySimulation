#pragma once
#include "CudaTreeNode.hpp"

OctreePtr CudaTreeNode::createCudaPtr() const
{
	CudaTreeNodeFlat* nodesCuda;
	size_t sizeOfMem = m_nodes->size() * sizeof(CudaTreeNodeFlat);
	//std::cout << "Cb nodes de l'octree cuda: " << m_nodes->size() << std::endl;
	hipMalloc((void**)&nodesCuda, sizeOfMem);
	hipMemcpy(nodesCuda, m_nodes->data(), sizeOfMem, hipMemcpyHostToDevice);

	size_t sizeOfMemLastNode = m_lastNodes->size() * sizeof(CudaLastTreeNodeFlat);
	CudaLastTreeNodeFlat* lastNodesCuda;
	//std::cout << "Cb dernier  nodes de l'octree cuda: " << m_lastNodes->size() << std::endl;
	hipMalloc((void**)&lastNodesCuda, sizeOfMemLastNode);
	hipMemcpy(lastNodesCuda, m_lastNodes->data(), sizeOfMemLastNode, hipMemcpyHostToDevice);
	return OctreePtr{nodesCuda, lastNodesCuda };
}

void CudaTreeNode::reset(const Bbox& bbox)
{
	SimplifiedTreeNode<8>::reset(bbox);
	m_nodeIdx =  0u;
	m_nodes->clear();
	m_nodes->push_back(
		CudaTreeNodeFlat{ PosLy(), 0.f, m_bbox.size * 2.f / s_theta }
	);
}

unsigned char CudaTreeNode::getIndex(const PosLy& pos) const
{
	unsigned char idx{ 0u };

	if (pos.x > m_bbox.center.x)
	{
		idx |= 1;
	}

	if (pos.y > m_bbox.center.y)
	{
		idx |= 2;
	}

	if (pos.z > m_bbox.center.z)
	{
		idx |= 4;
	}
	return idx;
}

Bbox CudaTreeNode::getSubBbox(unsigned char id) const
{
	return Bbox{
		{m_bbox.center.x + m_bbox.size * ((id & 1) ? 0.5f : -0.5f),
		 m_bbox.center.y + m_bbox.size * ((id & 2) ? 0.5f : -0.5f),
		 m_bbox.center.z + m_bbox.size * ((id & 4) ? 0.5f : -0.5f) },
	m_bbox.size / 2.f
	};
}

SimplifiedTreeNode<8>* CudaTreeNode::createChild(const Bbox& bbox, unsigned char idx) const
{
	CudaTreeNodeFlat nodeCuda{};
	nodeCuda.barnesHutCst = bbox.size * 2.f / s_theta;
	nodeCuda.centerOfMass = Vec3();
	nodeCuda.mass = 0.f;

	std::lock_guard<std::mutex> lock(*m_mutex.get());
	size_t childIdx = m_nodes->size();
	m_nodes->push_back(nodeCuda);
	m_nodes->at(m_nodeIdx).setChildIdx(idx, static_cast<int64_t>(childIdx));
	return new CudaTreeNode(m_mutex, m_nodes, m_lastNodes, childIdx, bbox);
}

ITreeNode* CudaTreeNode::createLastTreeNode(const PosLy& pos, MassMs mass, unsigned char idx) const
{
	CudaLastTreeNode* lastNode = new CudaLastTreeNode(m_mutex, m_lastNodes, pos, mass);
	int64_t childIdx = -static_cast<int64_t>(lastNode->getIndex()) - 1;

	std::lock_guard<std::mutex> lock(*m_mutex.get());
	m_nodes->at(m_nodeIdx).setChildIdx(idx, childIdx);
	return lastNode;
}