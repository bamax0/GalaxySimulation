#include "hip/hip_runtime.h"
#include "Algorithm/CudaSimpleAlgorithm.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void computeForces(Star* stars, size_t nbStars, Vec3* forces)
{
	size_t id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= nbStars) return;
	Vec3 f{ 0.f, 0.f, 0.f };
	for (size_t i{ 0u }; i < nbStars; ++i)
	{
		if (id != i)
		{
			f += computeForce(stars[id], stars[i]);
		}
	}
	Star blackHole;
	blackHole.position = { 0, 0, 0 };
	blackHole.mass = 2e6;
	f += computeForce(stars[id], blackHole);
	forces[id] = f;
	return;
}

__global__ void updatePositions(Star* stars, size_t nbStars, TimeS dt, Vec3* forces)
{
	size_t id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= nbStars) return;
	leapFrog(stars[id], getAcceleration(stars[id], forces[id]), dt);
}

GalaxySim::CudaSimpleAlgorithm::~CudaSimpleAlgorithm()
{
	hipFree(m_gpuStarsPtr);
	hipFree(m_gpuForcesPtr);
}

void GalaxySim::CudaSimpleAlgorithm::initWithGPU(size_t nbStars, Star* stars)
{
	m_nbStars = nbStars;
	m_gpuStarsPtr = stars;
	hipMalloc((void**)&m_gpuForcesPtr, m_nbStars * sizeof(Vec3));
}

void GalaxySim::CudaSimpleAlgorithm::init(const std::vector<Star>& stars)
{
	if (m_gpuStarsPtr != nullptr)
	{
		hipFree(m_gpuStarsPtr);
	}
	if (m_gpuForcesPtr != nullptr)
	{
		hipFree(m_gpuForcesPtr);
	}
	m_nbStars = stars.size();
	size_t sizeOfMem = m_nbStars * sizeof(Star);
	hipMalloc((void**)&m_gpuStarsPtr, sizeOfMem);
	hipMalloc((void**)&m_gpuForcesPtr, m_nbStars * sizeof(Vec3));
	hipMemcpy(m_gpuStarsPtr, stars.data(), sizeOfMem, hipMemcpyHostToDevice);
}

void GalaxySim::CudaSimpleAlgorithm::next(TimeY dt)
{
	int threadsPerBlock = 128;
	int blocksPerGrid = (m_nbStars + threadsPerBlock-1) / threadsPerBlock;

	// std::cout << m_nbStars << std::endl;
	//std::cout << "D�but calcule force" << std::endl;
	computeForces<<<blocksPerGrid, threadsPerBlock>>>(m_gpuStarsPtr, m_nbStars, m_gpuForcesPtr);

	//std::cout << "Fin calcule force" << std::endl;
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Kernel launch failed: %s\n", hipGetErrorString(err));
	}
	hipDeviceSynchronize();	
	hipError_t err3 = hipGetLastError();
	if (err3 != hipSuccess) {
		printf("Kernel launch failed3: %s\n", hipGetErrorString(err3));
	}
	updatePositions<<<blocksPerGrid, threadsPerBlock>>>(m_gpuStarsPtr, m_nbStars, dt, m_gpuForcesPtr);
}

std::vector<Star> GalaxySim::CudaSimpleAlgorithm::getStars() const
{
	std::vector<Star> stars(m_nbStars, Star{});
	hipMemcpy(stars.data(), m_gpuStarsPtr, m_nbStars * sizeof(Star), hipMemcpyDeviceToHost);
	hipError_t err3 = hipGetLastError();
	if (err3 != hipSuccess) {
		printf("Erreure a la r�cup�ration des �toile: %s\n", hipGetErrorString(err3));
	}
	return stars;
}

const Star* GalaxySim::CudaSimpleAlgorithm::getCudaStars() const
{
	return m_gpuStarsPtr;
}
